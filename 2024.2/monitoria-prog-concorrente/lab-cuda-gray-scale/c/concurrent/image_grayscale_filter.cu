#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "../lib/stb_image.h"
#include "../lib/stb_image_write.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#define BLOCK_SIZE_1D 1024

__global__ void convertToGrayscale(unsigned char *originalImage, unsigned char *grayscaleImage, int width, int height)
{

  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  // Convert current pixel to grayscale
  if (x < width && y < height)
  {
    int idx = (y * width + x) * 3;
    unsigned char r = originalImage[idx];
    unsigned char g = originalImage[idx + 1];
    unsigned char b = originalImage[idx + 2];

    // Apply grayscale formula
    grayscaleImage[y * width + x] =
        (unsigned char)(0.299 * r + 0.587 * g + 0.114 * b);
  }
}

/**
 * Applies a grayscale filter to an image.
 *
 * @param inputPath Path to the input image
 * @param outputPath Path to the output image
 * @param kernelSize Size of the mean kernel
 */
void applyGrayscaleFilter(const char *inputPath, const char *outputPath)
{
  // Load the input image
  int width, height, channels;
  unsigned char *hostOriginalImage =
      stbi_load(inputPath, &width, &height, &channels, 3);
  if (!hostOriginalImage)
  {
    fprintf(stderr, "Error: Unable to load image %s\n", inputPath);
    exit(1);
  }

  // Allocate memory for the original image in the device
  unsigned char *deviceOriginalImage;
  hipMalloc(&deviceOriginalImage, width * height * 3 * sizeof(unsigned char)); // 3 bytes per pixel

  // Copy the original image into the device
  hipMemcpy(deviceOriginalImage, hostOriginalImage,
             width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice);

  // Allocate device memory to store the grayscale image
  unsigned char *deviceGrayscaleImage;
  hipMalloc(&deviceGrayscaleImage, width * height * sizeof(unsigned char));

  dim3 blockSize(16, 16); // 16x16 threads per block
  dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  convertToGrayscale<<<gridSize, blockSize>>>(deviceOriginalImage, deviceGrayscaleImage, width, height);
  hipDeviceSynchronize();
  hipEventRecord(stop);

  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Kernel execution time: %f ms\n", milliseconds);

  // Allocate memory for the grayscale image in the host
  unsigned char *hostGrayscaleImage = (unsigned char *)malloc(width * height * sizeof(unsigned char));

  hipMemcpy(hostGrayscaleImage, deviceGrayscaleImage,
             width * height * sizeof(unsigned char), hipMemcpyDeviceToHost);

  // Save the grayscale image
  if (!stbi_write_jpg(outputPath, width, height, 1, hostGrayscaleImage, 100))
  {
    fprintf(stderr, "Error: Unable to save image %s\n", outputPath);
    free(hostGrayscaleImage);
    stbi_image_free(hostOriginalImage);
    exit(1);
  }

  // Clean up
  free(hostGrayscaleImage);
  stbi_image_free(hostOriginalImage);
  hipFree(deviceOriginalImage);
  hipFree(deviceGrayscaleImage);
}

/**
 * Main function for demonstration.
 *
 * Usage: ./script <input_file>
 */
int main(int argc, char *argv[])
{

    char cwd[PATH_MAX];
    if (getcwd(cwd, sizeof(cwd)) != NULL) {
        printf("Current working directory: %s\n", cwd);
    } else {
        perror("getcwd() error");
    }

  if (argc < 2)
  {
    fprintf(stderr, "Usage: %s <input_file>\n", argv[0]);
    return 1;
  }

  const char *inputFile = argv[1];
  const char *outputFile = "grayscale_output.jpg";

  applyGrayscaleFilter(inputFile, outputFile);

  printf("Grayscale image saved to %s\n", outputFile);

  return 0;
}